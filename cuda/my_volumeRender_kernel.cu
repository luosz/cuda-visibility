#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple 3D volume renderer

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include <iostream>
using namespace std;

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture

texture<float, hipTextureType3D, hipReadModeElementType>  volumeTexIn;
surface<void, 3>                                    volumeTexOut;
hipArray *d_visibilityArray = 0;

__device__ float *visVolume = NULL;
__device__ int *countVolume = NULL;
//__device__ __managed__ hipExtent sizeOfVolume;// = make_hipExtent(32, 32, 32);

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__device__ void addVisibility(float value, float3 pos)
{
	int w, h, d;
	w = h = d = 32;
	int x = (int)((pos.x + 1) * 0.5f * w + 0.5f);
	x = (x >= w) ? (w - 1) : x;
	int y = (int)((pos.y + 1) * 0.5f * h + 0.5f);
	y = (y >= h) ? (h - 1) : y;
	int z = (int)((pos.z + 1) * 0.5f * d + 0.5f);
	z = (z >= d) ? (d - 1) : z;

	int index = z*w*h + y*w + x;
	
	//countVolume[index] += 1;
	//visVolume[index] += value;
	atomicAdd((countVolume + index), 1);
	atomicAdd((visVolume + index), value);
}

__global__ void
surf_write(float *data, hipExtent volumeSize)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;

	if (x >= volumeSize.width || y >= volumeSize.height || z >= volumeSize.depth)
	{
		return;
	}
	float output = data[z*(volumeSize.width*volumeSize.height) + y*(volumeSize.width) + x];
	// surface writes need byte offsets for x!
	surf3Dwrite(output, volumeTexOut, x * sizeof(float), y, z);
}

__global__ void
tex_read(float x, float y, float z) {
	printf("x: %f, y: %f, z:%f, val: %f\n", x, y, z, tex3D(volumeTexIn, x, y, z));
}

__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale)
{
    const int maxSteps = 500;
    const float tstep = 0.01f;
    const float opacityThreshold = 0.95f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;

    for (int i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates
        float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
        //sample *= 64.0f;    // scale for 10-bit data

        // lookup in transfer function texture
        float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
        col.w *= density;

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);

        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        // exit early if opaque
        if (sum.w > opacityThreshold)
            break;

        t += tstep;

        if (t > tfar) break;

        pos += step;
    }

    sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

__global__ void
d_visibility(uint *d_output, uint imageW, uint imageH,
	float density, float brightness,
	float transferOffset, float transferScale)
{
	const int maxSteps = 500;
	const float tstep = 0.01f;
	const float opacityThreshold = 0.95f;
	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
	const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= imageW) || (y >= imageH)) return;

	float u = (x / (float)imageW)*2.0f - 1.0f;
	float v = (y / (float)imageH)*2.0f - 1.0f;

	// calculate eye ray in world space
	Ray eyeRay;
	eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	eyeRay.d = normalize(make_float3(u, v, -2.0f));
	eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

	// find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

	if (!hit) return;

	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

										// march along ray from front to back, accumulating color
	float4 sum = make_float4(0.0f);
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	for (int i = 0; i<maxSteps; i++)
	{
		// read from 3D texture
		// remap position to [0, 1] coordinates
		float sample = tex3D(tex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
		//sample *= 64.0f;    // scale for 10-bit data

		// lookup in transfer function texture
		float4 col = tex1D(transferTex, (sample - transferOffset)*transferScale);
		col.w *= density;

		// "under" operator for back-to-front blending
		//sum = lerp(sum, col, col.w);

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		// "over" operator for front-to-back blending
		sum = sum + col*(1.0f - sum.w);

		addVisibility(sum.w, pos);

		// exit early if opaque
		if (sum.w > opacityThreshold)
			break;

		t += tstep;

		if (t > tfar) break;

		pos += step;
	}

	sum *= brightness;

	// write output color
	d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

extern "C"
void initCuda(void *h_volume, hipExtent volumeSize)
{
	auto len = volumeSize.width * volumeSize.height * volumeSize.depth;
	auto cube = malloc(sizeof(float) * len);
	memset(cube, 0, sizeof(float) * len);
	printf("%g\n", *((float*)cube+len-1));

	//sizeOfVolume = volumeSize;
	//printf("%d %d %d\n", sizeOfVolume.width, sizeOfVolume.height, sizeOfVolume.depth);
	//printf("%d %d %d\n", volumeSize.width, volumeSize.height, volumeSize.depth);

	//auto len = volumeSize.width * volumeSize.height * volumeSize.depth;
	checkCudaErrors(hipMallocManaged(&visVolume, sizeof(float) * len));
	checkCudaErrors(hipMallocManaged(&countVolume, sizeof(int) * len));
	printf("%g\n", *(visVolume + 1));
	printf("%d\n", *(countVolume + 1));

    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

	std::cout << "channelDesc\t" << channelDesc.x << "\t" << channelDesc.y << "\t" << channelDesc.z << "\t" << channelDesc.w << "\t" << channelDesc.f << std::endl;
	std::cout << "volumeSize\t" << volumeSize.width << "\t" << volumeSize.height << "\t" << volumeSize.depth << std::endl;

	auto channelDesc0 = hipCreateChannelDesc<float>();
	checkCudaErrors(hipMalloc3DArray(&d_visibilityArray, &channelDesc0, volumeSize, hipArraySurfaceLoadStore));
	std::cout << "channelDesc\t" << channelDesc0.x << "\t" << channelDesc0.y << "\t" << channelDesc0.z << "\t" << channelDesc0.w << "\t" << channelDesc0.f << std::endl;

	auto ptr = make_hipPitchedPtr(d_visibilityArray, volumeSize.width * sizeof(VolumeType), volumeSize.width, volumeSize.height);
	//checkCudaErrors(hipMemset3D(ptr, 0, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

	auto copyParams2 = copyParams;
	copyParams2.srcPtr = make_hipPitchedPtr(cube, volumeSize.width * sizeof(float), volumeSize.width, volumeSize.height);
	copyParams2.dstArray = d_visibilityArray;
	checkCudaErrors(hipMemcpy3D(&copyParams2));

	volumeTexIn.filterMode = hipFilterModeLinear;
	checkCudaErrors(cudaBindSurfaceToArray(volumeTexOut, d_visibilityArray));

	dim3 blockSize(8, 8, 8);
	dim3 gridSize((volumeSize.width + 7) / 8, (volumeSize.height + 7) / 8, (volumeSize.depth + 7) / 8);
	//surf_write << <gridSize, blockSize >> >((float *)cube, volumeSize);

	checkCudaErrors(hipBindTextureToArray(volumeTexIn, d_visibilityArray));

	//tex_read << <1, 1 >> >(1.5, 1.5, 1.5);

	//checkCudaErrors(hipDeviceSynchronize());

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

    // create transfer function texture
    float4 transferFunc[] =
    {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}

extern "C"
void freeCudaBuffers()
{
    checkCudaErrors(hipFreeArray(d_volumeArray));
	checkCudaErrors(hipFreeArray(d_transferFuncArray));
	checkCudaErrors(hipFreeArray(d_visibilityArray));
	checkCudaErrors(hipFree(visVolume));
	checkCudaErrors(hipFree(countVolume));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale)
{
    //d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density, brightness, transferOffset, transferScale);

	d_visibility << <gridSize, blockSize >> >(d_output, imageW, imageH, density, brightness, transferOffset, transferScale);
	hipDeviceSynchronize();
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
