#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple 3D volume renderer

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include <iostream>
using namespace std;

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture

texture<float, hipTextureType3D, hipReadModeElementType>  volumeTexIn;
surface<void, 3>                                    volumeTexOut;
hipArray *d_visibilityArray = 0;

__device__ __managed__ float *visVolume = NULL;
__device__ __managed__ int *countVolume = NULL;
__device__ __managed__ hipExtent sizeOfVolume;// = make_hipExtent(32, 32, 32);
typedef float VisibilityType;
texture<VisibilityType, 3, hipReadModeElementType> visTex;         // 3D texture
//texture<VisibilityType, 3, hipReadModeNormalizedFloat> visTex;         // 3D texture

// save visibility
bool save_visibility = false;

extern "C" void set_save(bool value)
{
	save_visibility = value;
	printf("set save %s\n", save_visibility ?"true":"false");
}

extern "C" bool get_save()
{
	//printf("get save %s\n", save_visibility ? "true" : "false");
	return save_visibility;
}

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__device__ void addVisibility(float value, float3 pos)
{
	int w = sizeOfVolume.width, h = sizeOfVolume.height, d = sizeOfVolume.depth;
	//w = h = d = 32;
	int x = (int)((pos.x + 1) * 0.5f * w + 0.5f);
	x = (x >= w) ? (w - 1) : x;
	int y = (int)((pos.y + 1) * 0.5f * h + 0.5f);
	y = (y >= h) ? (h - 1) : y;
	int z = (int)((pos.z + 1) * 0.5f * d + 0.5f);
	z = (z >= d) ? (d - 1) : z;

	int index = z*w*h + y*w + x;
	
	//countVolume[index] += 1;
	//visVolume[index] += value;
	atomicAdd((countVolume + index), 1);
	atomicAdd((visVolume + index), value);
	//printf("atomicAdd %d \t %g \n", countVolume[index], visVolume[index]);
}

__global__ void
surf_write(float *data, hipExtent volumeSize)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;

	if (x >= volumeSize.width || y >= volumeSize.height || z >= volumeSize.depth)
	{
		return;
	}
	float output = data[z*(volumeSize.width*volumeSize.height) + y*(volumeSize.width) + x];
	// surface writes need byte offsets for x!
	surf3Dwrite(output, volumeTexOut, x * sizeof(float), y, z);
}

__global__ void
tex_read(float x, float y, float z) {
	printf("x: %f, y: %f, z:%f, val: %f\n", x, y, z, tex3D(volumeTexIn, x, y, z));
}

__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale)
{
    const int maxSteps = 500;
    const float tstep = 0.01f;
    const float opacityThreshold = 0.95f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;

    for (int i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates
        float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
        //sample *= 64.0f;    // scale for 10-bit data

        // lookup in transfer function texture
        float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
        col.w *= density;

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);

        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        // exit early if opaque
        if (sum.w > opacityThreshold)
            break;

        t += tstep;

        if (t > tfar) break;

        pos += step;
    }

    sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

__global__ void
d_visibility(uint *d_output, uint imageW, uint imageH,
	float density, float brightness,
	float transferOffset, float transferScale)
{
	const int maxSteps = 500;
	const float tstep = 0.01f;
	const float opacityThreshold = 0.95f;
	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
	const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= imageW) || (y >= imageH)) return;

	float u = (x / (float)imageW)*2.0f - 1.0f;
	float v = (y / (float)imageH)*2.0f - 1.0f;

	// calculate eye ray in world space
	Ray eyeRay;
	eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	eyeRay.d = normalize(make_float3(u, v, -2.0f));
	eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

	// find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

	if (!hit) return;

	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

										// march along ray from front to back, accumulating color
	float4 sum = make_float4(0.0f);
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	for (int i = 0; i<maxSteps; i++)
	{
		// read from 3D texture
		// remap position to [0, 1] coordinates
		float sample = tex3D(tex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
		//sample *= 64.0f;    // scale for 10-bit data

		// lookup in transfer function texture
		float4 col = tex1D(transferTex, (sample - transferOffset)*transferScale);
		col.w *= density;

		// "under" operator for back-to-front blending
		//sum = lerp(sum, col, col.w);

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		// "over" operator for front-to-back blending

		float sumw = sum.w;
		sum = sum + col*(1.0f - sum.w);

		addVisibility(sum.w - sumw, pos);

		// exit early if opaque
		if (sum.w > opacityThreshold)
			break;

		t += tstep;

		if (t > tfar) break;

		pos += step;
	}

	sum *= brightness;

	// write output color
	d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

__global__ void
d_renderVisibility(uint *d_output, uint imageW, uint imageH,
	float density, float brightness,
	float transferOffset, float transferScale)
{
	const int maxSteps = 500;
	const float tstep = 0.01f;
	const float opacityThreshold = 0.95f;
	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
	const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= imageW) || (y >= imageH)) return;

	float u = (x / (float)imageW)*2.0f - 1.0f;
	float v = (y / (float)imageH)*2.0f - 1.0f;

	// calculate eye ray in world space
	Ray eyeRay;
	eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	eyeRay.d = normalize(make_float3(u, v, -2.0f));
	eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

	// find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

	if (!hit) return;

	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

										// march along ray from front to back, accumulating color
	float4 sum = make_float4(0.0f);
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	for (int i = 0; i<maxSteps; i++)
	{
		// read from 3D texture
		// remap position to [0, 1] coordinates
		float sample = tex3D(visTex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
		//sample *= 64.0f;    // scale for 10-bit data

		// lookup in transfer function texture
		//float4 col = tex1D(transferTex, (sample - transferOffset)*transferScale);
		float4 col = make_float4(sample, sample, sample, sample);
		col.w *= density;

		// "under" operator for back-to-front blending
		//sum = lerp(sum, col, col.w);

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		// "over" operator for front-to-back blending
		sum = sum + col*(1.0f - sum.w);

		// exit early if opaque
		if (sum.w > opacityThreshold)
			break;

		t += tstep;

		if (t > tfar) break;

		pos += step;
	}

	sum *= brightness;

	// write output color
	d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

extern "C"
void initCuda(void *h_volume, hipExtent volumeSize)
{
	auto len = volumeSize.width * volumeSize.height * volumeSize.depth;
	auto cube = malloc(sizeof(float) * len);
	memset(cube, 0, sizeof(float) * len);
	printf("%g\n", *((float*)cube+len-1));

	sizeOfVolume = volumeSize;
	printf("sizeOfVolume \t %d %d %d\n", sizeOfVolume.width, sizeOfVolume.height, sizeOfVolume.depth);
	printf("volumeSize \t %d %d %d\n", volumeSize.width, volumeSize.height, volumeSize.depth);

	//auto len = volumeSize.width * volumeSize.height * volumeSize.depth;
	checkCudaErrors(hipMallocManaged(&visVolume, sizeof(float) * len));
	checkCudaErrors(hipMallocManaged(&countVolume, sizeof(int) * len));
	printf("%g\n", *(visVolume + 1));
	printf("%d\n", *(countVolume + 1));

    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));
	std::cout << "channelDesc \t" << channelDesc.x << "\t" << channelDesc.y << "\t" << channelDesc.z << "\t" << channelDesc.w << "\t" << channelDesc.f << std::endl;

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

    // create transfer function texture
    float4 transferFunc[] =
    {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}

extern "C"
void freeCudaBuffers()
{
    checkCudaErrors(hipFreeArray(d_volumeArray));
	checkCudaErrors(hipFreeArray(d_transferFuncArray));
	checkCudaErrors(hipFreeArray(d_visibilityArray));
	checkCudaErrors(hipFree(visVolume));
	checkCudaErrors(hipFree(countVolume));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale)
{
    //d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density, brightness, transferOffset, transferScale);

	auto len = sizeOfVolume.width * sizeOfVolume.height * sizeOfVolume.depth;
	//auto cube = malloc(sizeof(float) * len);
	//memset(visVolume, 0, sizeof(VisibilityType) * len);
	hipMemset(visVolume, 0, sizeof(VisibilityType) * len);

	d_visibility << <gridSize, blockSize >> >(d_output, imageW, imageH, density, brightness, transferOffset, transferScale);
	hipDeviceSynchronize();

	hipChannelFormatDesc channelDesc0 = hipCreateChannelDesc<VisibilityType>();
	//checkCudaErrors(hipMalloc3DArray(&d_visibilityArray, &channelDesc0, sizeOfVolume, hipArraySurfaceLoadStore));
	checkCudaErrors(hipMalloc3DArray(&d_visibilityArray, &channelDesc0, sizeOfVolume));
	//std::cout << "channelDesc0 \t" << channelDesc0.x << "\t" << channelDesc0.y << "\t" << channelDesc0.z << "\t" << channelDesc0.w << "\t" << channelDesc0.f << std::endl;

	// copy data to 3D array
	hipMemcpy3DParms copyParams2 = { 0 };
	copyParams2.srcPtr = make_hipPitchedPtr(visVolume, sizeOfVolume.width * sizeof(VisibilityType), sizeOfVolume.width, sizeOfVolume.height);
	copyParams2.dstArray = d_visibilityArray;
	copyParams2.extent = sizeOfVolume;
	copyParams2.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams2));

	//checkCudaErrors(cudaMemcpyToArrayAsync(d_visibilityArray, 0, 0, countVolume, volumeSize.width*volumeSize.height*sizeof(int), hipMemcpyHostToDevice));

	// set texture parameters
	visTex.normalized = true;                      // access with normalized texture coordinates
	visTex.filterMode = hipFilterModeLinear;      // linear interpolation
	//visTex.filterMode = hipFilterModePoint;      // linear interpolation
	visTex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	visTex.addressMode[1] = hipAddressModeClamp;

	checkCudaErrors(hipBindTextureToArray(visTex, d_visibilityArray, channelDesc0));

	d_renderVisibility << <gridSize, blockSize >> >(d_output, imageW, imageH, density, brightness, transferOffset, transferScale);

	hipDeviceSynchronize();

	if (get_save())
	{
		set_save(false);
		printf("save visibility to visibility.raw.\n");

		auto fp = fopen("visibility.raw", "wb");
		fwrite(visVolume, sizeof(VisibilityType), len, fp);
		fclose(fp);
	}
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
