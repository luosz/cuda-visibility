#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 // Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <iostream>
#include <stdio.h>
#include "define.cuh"
using namespace std;

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture

texture<float, hipTextureType3D, hipReadModeElementType>  volumeTexIn;
surface<void, 3>                                    volumeTexOut;
hipArray *d_visibilityArray = 0;

__device__ __managed__ VolumeType *raw_volume = NULL;
__device__ __managed__ char *volume_file = NULL;
__device__ __managed__ float *visVolume = NULL;
__device__ __managed__ int *countVolume = NULL;
__device__ __managed__ float *depthVolume = NULL;
__device__ __managed__ hipExtent sizeOfVolume;// = make_hipExtent(32, 32, 32);
typedef float VisibilityType;
texture<VisibilityType, 3, hipReadModeElementType> visTex;         // 3D texture
texture<VolumeType, 3, hipReadModeElementType> volTex;         // 3D texture
//texture<VisibilityType, 3, hipReadModeNormalizedFloat> visTex;         // 3D texture

const int BIN_COUNT = D_BIN_COUNT;
__device__ __managed__ float histogram[BIN_COUNT] = {0};
__device__ __managed__ float histogram2[BIN_COUNT] = { 0 };
__device__ __managed__ float histogram3[BIN_COUNT] = { 0 };
__device__ __managed__ float histogram4[BIN_COUNT] = { 0 };
__device__ __managed__ float4 tf_array[BIN_COUNT] = { 0 };
__device__ __managed__ float4 tf_array0[BIN_COUNT] = { 0 };
__device__ __managed__ int radius = D_RADIUS;
__device__ __managed__ float g5[R5*R5*R5] = { 0 };
__device__ __managed__ float g9[R9*R9*R9] = { 0 };
__device__ __managed__ float *saliencyVolume = NULL;
__device__ __managed__ float *vwsVolume = NULL;
bool saliency_once = false;
__device__ __managed__ unsigned char *featureVolume = NULL;
__device__ __managed__ int feature_number = 0;
__device__ __managed__ float feature_array[BIN_COUNT] = { 0 };
__device__ __managed__ float feature_vws_array[BIN_COUNT] = { 0 };

// GUI settings
//float g_SelectedColor[] = { 1.f,1.f,0.f,1.f };
float g_SelectedColor[] = { D_RGBA[0], D_RGBA[1], D_RGBA[2], D_RGBA[3] };
bool g_ApplyAlpha = D_APPLY_ALPHA;
bool g_ApplyColor = D_APPLY_COLOR;

// apply, save and discard operations
bool apply_blend = false;
bool discard_table = false;
bool save_histogram = false;
bool gaussian_histogram = false;
bool backup_table = false;

extern "C" float4 rgb_to_lch(float4 rgba);
extern "C" int iDivUp(int a, int b);

extern "C" VolumeType * get_raw_volume()
{
	return raw_volume;
}

extern "C" unsigned char * get_feature_volume()
{
	return featureVolume;
}

extern "C" float * get_vws_volume()
{
	return vwsVolume;
}

extern "C" int get_feature_number()
{
	return feature_number;
}

extern "C" void set_feature_number(int val)
{
	feature_number = val;
}

extern "C" float * get_feature_array()
{
	return feature_array;
}

extern "C" float * get_feature_vws_array()
{
	return feature_vws_array;
}

typedef float(*Pointer)[4];
extern "C" Pointer get_SelectedColor()
{
	return &g_SelectedColor;
}

extern "C" void set_SelectedColor(float r, float g, float b)
{
	g_SelectedColor[0] = r;
	g_SelectedColor[1] = g;
	g_SelectedColor[2] = b;
}

extern "C" bool* get_ApplyColor()
{
	return &g_ApplyColor;
}

extern "C" bool* get_ApplyAlpha()
{
	return &g_ApplyAlpha;
}

extern "C" int get_region_size()
{
	return radius;
}

extern "C" float4* get_tf_array()
{
	return tf_array;
}

extern "C" void backup_tf()
{
	memcpy(tf_array0, tf_array, sizeof(tf_array));
}

extern "C" void bind_tf_texture()
{
	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
	hipArray *d_transferFuncArray;
	checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(tf_array) / sizeof(float4), 1));
	checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, tf_array, sizeof(tf_array), hipMemcpyHostToDevice));
	// Bind the array to the texture
	checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}

extern "C" void restore_tf()
{
	memcpy(tf_array, tf_array0, sizeof(tf_array));
	bind_tf_texture();
}

extern "C" int get_bin_count()
{
	return BIN_COUNT;
}

extern "C" bool get_save()
{
	return save_histogram;
}

extern "C" void set_save(bool value)
{
	save_histogram = value;
	printf("set save %s\n", save_histogram ?"true":"false");
}

extern "C" bool get_apply()
{
	return apply_blend;
}

extern "C" void set_apply(bool value)
{
	apply_blend = value;
	printf("set apply %s\n", apply_blend ? "true" : "false");
}

extern "C" bool get_discard()
{
	return discard_table;
}

extern "C" void set_discard(bool value)
{
	discard_table = value;
	printf("set discard %s\n", discard_table ? "true" : "false");
}

extern "C" bool get_gaussian()
{
	return gaussian_histogram;
}

extern "C" void set_gaussian(bool value)
{
	gaussian_histogram = value;
	printf("set gaussian %s\n", gaussian_histogram ? "true" : "false");
}

extern "C" bool get_backup()
{
	return backup_table;
}

extern "C" void set_backup(bool value)
{
	backup_table = value;
	printf("set backup %s\n", backup_table ? "true" : "false");
}

extern "C" void set_volume_file(const char *file, int n)
{
	n = n + 1;
	if (!volume_file)
	{
		checkCudaErrors(hipMallocManaged(&volume_file, sizeof(float) * n));
	}
	memcpy(volume_file, file, n);
}

extern "C" void blend_tf(float3 color)
{
	float hist[BIN_COUNT];
	float max = 0;
	for (int i = 0; i < BIN_COUNT; i++)
	{
		if (max < histogram2[i])
		{
			max = histogram2[i];
		}
	}

	for (int i = 0; i < BIN_COUNT; i++)
	{
		hist[i] = histogram2[i] / max;
	}
	for (int i = 0; i < BIN_COUNT; i++)
	{
		auto c = make_float3(tf_array[i].x, tf_array[i].y, tf_array[i].z);
		auto c2 = lerp(c, color, hist[i]);
		if (hist[i] > 0.5)
		{
			printf("%g r %g %g g %g %g b %g %g \n", i/(float)BIN_COUNT, tf_array[i].x, c2.x, tf_array[i].y, c2.y, tf_array[i].z, c2.z);
		}
		tf_array[i].x = c2.x;
		tf_array[i].y = c2.y;
		tf_array[i].z = c2.z;
	}

	bind_tf_texture();
}

extern "C" void blend_tf_rgba(float3 color)
{
	float hist[BIN_COUNT], hist2[BIN_COUNT];
	float sum = 0;
	for (int i = 0; i < BIN_COUNT; i++)
	{
		sum += histogram[i];
	}
	for (int i = 0; i < BIN_COUNT; i++)
	{
		hist[i] = histogram[i] / sum;
	}
	float sum2 = 0;
	for (int i = 0; i < BIN_COUNT; i++)
	{
		sum2 += histogram2[i];
	}
	for (int i = 0; i < BIN_COUNT; i++)
	{
		hist2[i] = histogram2[i] / sum2;
	}
	float max = 0;
	for (int i = 0; i < BIN_COUNT; i++)
	{
		histogram3[i] = hist2[i] - hist[i];
		auto m = fabsf(histogram3[i]);
		max = max < m ? m : max;
	}

	// apply Gaussian filter to relateive visibility histogram
	memcpy(histogram4, histogram3, BIN_COUNT * sizeof(float));
	gaussian(histogram4, BIN_COUNT);

	// normalize histogram3
	for (int i = 0; i < BIN_COUNT; i++)
	{
		histogram3[i] /= max;
	}

	// normalize histogram4
	max = 0;
	for (int i = 0; i < BIN_COUNT; i++)
	{
		auto m = fabsf(histogram4[i]);
		max = max < m ? m : max;
	}
	for (int i = 0; i < BIN_COUNT; i++)
	{
		histogram4[i] /= max;
	}

	if (g_ApplyColor)
	{
		for (int i = 0; i < BIN_COUNT; i++)
		{
			auto c = make_float3(tf_array[i].x, tf_array[i].y, tf_array[i].z);
			auto t = histogram3[i] > 0 ? histogram3[i] : 0;
			auto c2 = lerp(c, color, t);
			tf_array[i].x = c2.x;
			tf_array[i].y = c2.y;
			tf_array[i].z = c2.z;
		}
	}

	if (g_ApplyAlpha)
	{
		for (int i = 0; i < BIN_COUNT; i++)
		{
			tf_array[i].w = lerp(tf_array[i].w, histogram3[i] > 0 ? 1 : 0, fabsf(histogram3[i]));
		}
	}

	bind_tf_texture();
}

extern "C" void gaussian_tf(float3 color)
{
	float hist[BIN_COUNT], hist2[BIN_COUNT];
	float sum = 0;
	for (int i = 0; i < BIN_COUNT; i++)
	{
		sum += histogram[i];
	}
	for (int i = 0; i < BIN_COUNT; i++)
	{
		hist[i] = histogram[i] / sum;
	}
	float sum2 = 0;
	for (int i = 0; i < BIN_COUNT; i++)
	{
		sum2 += histogram2[i];
	}
	for (int i = 0; i < BIN_COUNT; i++)
	{
		hist2[i] = histogram2[i] / sum2;
	}
	float max = 0;
	for (int i = 0; i < BIN_COUNT; i++)
	{
		histogram3[i] = hist2[i] - hist[i];
		auto m = fabsf(histogram3[i]);
		max = max < m ? m : max;
	}

	// apply Gaussian filter to relateive visibility histogram
	memcpy(histogram4, histogram3, BIN_COUNT * sizeof(float));
	gaussian(histogram4, BIN_COUNT);

	// normalize histogram3
	for (int i = 0; i < BIN_COUNT; i++)
	{
		histogram3[i] /= max;
	}

	// normalize histogram4
	max = 0;
	for (int i = 0; i < BIN_COUNT; i++)
	{
		auto m = fabsf(histogram4[i]);
		max = max < m ? m : max;
	}
	for (int i = 0; i < BIN_COUNT; i++)
	{
		histogram4[i] /= max;
	}

	if (g_ApplyColor)
	{
		for (int i = 0; i < BIN_COUNT; i++)
		{
			auto c = make_float3(tf_array[i].x, tf_array[i].y, tf_array[i].z);
			auto t = histogram4[i] > 0 ? histogram4[i] : 0;
			auto c2 = lerp(c, color, t);
			tf_array[i].x = c2.x;
			tf_array[i].y = c2.y;
			tf_array[i].z = c2.z;
		}
	}

	if (g_ApplyAlpha)
	{
		for (int i = 0; i < BIN_COUNT; i++)
		{
			auto a = histogram4[i] > 0 ? 1 : 0;
			auto t = fabsf(histogram4[i]);
			tf_array[i].w = lerp(tf_array[i].w, a, t);
		}
	}

	bind_tf_texture();
}

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__device__ void addVisibility(float value, float3 pos, float depth)
{
	int w = sizeOfVolume.width, h = sizeOfVolume.height, d = sizeOfVolume.depth;
	//w = h = d = 32;
	//pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f
	int x = (int)((pos.x*0.5f + 0.5f) * w + 0.5f);
	x = (x >= w) ? (w - 1) : x;
	int y = (int)((pos.y*0.5f + 0.5f) * h + 0.5f);
	y = (y >= h) ? (h - 1) : y;
	int z = (int)((pos.z*0.5f + 0.5f) * d + 0.5f);
	z = (z >= d) ? (d - 1) : z;

	int index = z*w*h + y*w + x;
	
	atomicAdd((countVolume + index), 1);
	atomicAdd((visVolume + index), value);
	if (fabsf(depthVolume[index]) < 1e-6)
	{
		atomicAdd((depthVolume + index), depth);
	}
	
	//printf("atomicAdd %d \t %g \n", countVolume[index], visVolume[index]);

	float sample = tex3D(volTex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
	VolumeType intensity = (int)(sample + 0.5f);
	//float sample = tex3D(tex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
	//VolumeType intensity = (int)(sample*255 + 0.5f);
	atomicAdd((histogram + intensity), value);
}

__device__ void addVisibility2(float value, float3 pos)
{
	int w = sizeOfVolume.width, h = sizeOfVolume.height, d = sizeOfVolume.depth;
	//w = h = d = 32;
	//pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f
	int x = (int)((pos.x*0.5f + 0.5f) * w + 0.5f);
	x = (x >= w) ? (w - 1) : x;
	int y = (int)((pos.y*0.5f + 0.5f) * h + 0.5f);
	y = (y >= h) ? (h - 1) : y;
	int z = (int)((pos.z*0.5f + 0.5f) * d + 0.5f);
	z = (z >= d) ? (d - 1) : z;

	//int index = z*w*h + y*w + x;
	//atomicAdd((countVolume + index), 1);
	//atomicAdd((visVolume + index), value);

	float sample = tex3D(volTex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
	VolumeType intensity = (int)(sample + 0.5f);
	//float sample = tex3D(tex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
	//VolumeType intensity = (int)(sample*255 + 0.5f);
	atomicAdd((histogram2 + intensity), value);
}

__global__ void
surf_write(float *data, hipExtent volumeSize)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;

	if (x >= volumeSize.width || y >= volumeSize.height || z >= volumeSize.depth)
	{
		return;
	}
	float output = data[z*(volumeSize.width*volumeSize.height) + y*(volumeSize.width) + x];
	// surface writes need byte offsets for x!
	surf3Dwrite(output, volumeTexOut, x * sizeof(float), y, z);
}

__global__ void
tex_read(float x, float y, float z) {
	printf("x: %f, y: %f, z:%f, val: %f\n", x, y, z, tex3D(volumeTexIn, x, y, z));
}

__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale)
{
    const int maxSteps = 500;
    const float tstep = 0.01f;
    const float opacityThreshold = 0.95f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;

    for (int i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates
        float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
        //sample *= 64.0f;    // scale for 10-bit data

        // lookup in transfer function texture
        float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
        col.w *= density;

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);

        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        // exit early if opaque
        if (sum.w > opacityThreshold)
            break;

        t += tstep;

        if (t > tfar) break;

        pos += step;
    }

    sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

__global__ void
d_visibility(uint *d_output, uint imageW, uint imageH,
	float density, float brightness,
	float transferOffset, float transferScale)
{
	const int maxSteps = 500;
	const float tstep = 0.01f;
	const float opacityThreshold = 0.95f;
	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
	const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= imageW) || (y >= imageH)) return;

	float u = (x / (float)imageW)*2.0f - 1.0f;
	float v = (y / (float)imageH)*2.0f - 1.0f;

	// calculate eye ray in world space
	Ray eyeRay;
	eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	eyeRay.d = normalize(make_float3(u, v, -2.0f));
	eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

	// find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

	if (!hit) return;

	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

										// march along ray from front to back, accumulating color
	float4 sum = make_float4(0.0f);
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	for (int i = 0; i<maxSteps; i++)
	{
		// read from 3D texture
		// remap position to [0, 1] coordinates
		float sample = tex3D(tex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
		//sample *= 64.0f;    // scale for 10-bit data

		// lookup in transfer function texture
		float4 col = tex1D(transferTex, (sample - transferOffset)*transferScale);
		col.w *= density;

		// "under" operator for back-to-front blending
		//sum = lerp(sum, col, col.w);

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		// "over" operator for front-to-back blending

		float sumw = sum.w;
		sum = sum + col*(1.0f - sum.w);

		addVisibility(sum.w - sumw, pos, t - tnear);

		// exit early if opaque
		if (sum.w > opacityThreshold)
			break;

		t += tstep;

		if (t > tfar) break;

		pos += step;
	}

	sum *= brightness;

	// write output color
	d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

__global__ void
d_visibilityLocal(uint *d_output, uint imageW, uint imageH,
	float density, float brightness,
	float transferOffset, float transferScale, int2 loc)
{
	const int maxSteps = 500;
	const float tstep = 0.01f;
	const float opacityThreshold = 0.95f;
	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
	const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= imageW) || (y >= imageH)) return;

	float u = (x / (float)imageW)*2.0f - 1.0f;
	float v = (y / (float)imageH)*2.0f - 1.0f;

	// calculate eye ray in world space
	Ray eyeRay;
	eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	eyeRay.d = normalize(make_float3(u, v, -2.0f));
	eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

	// find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

	if (!hit) return;

	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

										// march along ray from front to back, accumulating color
	float4 sum = make_float4(0.0f);
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	for (int i = 0; i<maxSteps; i++)
	{
		// read from 3D texture
		// remap position to [0, 1] coordinates
		float sample = tex3D(tex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
		//sample *= 64.0f;    // scale for 10-bit data

		// lookup in transfer function texture
		float4 col = tex1D(transferTex, (sample - transferOffset)*transferScale);
		col.w *= density;

		// "under" operator for back-to-front blending
		//sum = lerp(sum, col, col.w);

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		// "over" operator for front-to-back blending

		float sumw = sum.w;
		sum = sum + col*(1.0f - sum.w);

		addVisibility(sum.w - sumw, pos, t- tnear);
		
		// calculate visibility for selected region
		if (fabsf(x - loc.x) <= radius && fabsf(y - loc.y) <= radius)
		{
			addVisibility2(sum.w - sumw, pos);
		}

		// exit early if opaque
		if (sum.w > opacityThreshold)
			break;

		t += tstep;

		if (t > tfar) break;

		pos += step;
	}
	sum *= brightness;

	// write output color
	d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

__global__ void
d_renderVisibility(uint *d_output, uint imageW, uint imageH,
	float density, float brightness,
	float transferOffset, float transferScale, int2 loc)
{
	const int maxSteps = 500;
	const float tstep = 0.01f;
	const float opacityThreshold = 0.95f;
	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
	const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= imageW) || (y >= imageH)) return;

	float u = (x / (float)imageW)*2.0f - 1.0f;
	float v = (y / (float)imageH)*2.0f - 1.0f;

	// calculate eye ray in world space
	Ray eyeRay;
	eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	eyeRay.d = normalize(make_float3(u, v, -2.0f));
	eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

	// find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

	if (!hit)
	{
		d_output[y*imageW + x] = rgbaFloatToInt(make_float4(1.0f, 1.0f, 1.0f, 0.0f));
		return;
	}

	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

	// march along ray from front to back, accumulating color
	float4 sum = make_float4(0.0f);
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	for (int i = 0; i<maxSteps; i++)
	{
		// read from 3D texture
		// remap position to [0, 1] coordinates
		float sample = tex3D(tex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
		//float vis = tex3D(visTex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
		//sample *= 64.0f;    // scale for 10-bit data

		// lookup in transfer function texture
		float4 col = tex1D(transferTex, (sample - transferOffset)*transferScale);
		//float4 col = make_float4(sample, sample, sample, sample);
		col.w *= density;
		//col.w /= vis;

		// "under" operator for back-to-front blending
		//sum = lerp(sum, col, col.w);

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		// "over" operator for front-to-back blending
		sum = sum + col*(1.0f - sum.w);

		// exit early if opaque
		if (sum.w > opacityThreshold)
			break;

		t += tstep;

		if (t > tfar) break;

		pos += step;
	}

	if (sum.w < 1.0f)
	{
		sum += make_float4(1.0f, 1.0f, 1.0f, 0.0f) * (1.0f - sum.w);
	}
	sum *= brightness;

	// draw selected region in inverted colors
	if (fabsf(x - loc.x) <= radius && fabsf(y - loc.y) <= radius)
	{
		auto w = sum.w;
		sum = make_float4(1, 1, 1, 1) - sum;
		sum.w = w;
	}

	// write output color
	d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

inline void load_gaussians()
{
	FILE *gf1 = fopen("gaussian_5_5_5.txt", "r");
	int n = R5*R5*R5;
	for (int i = 0; i < n; i++)
	{
		fscanf(gf1, "%g", &g5[i]);
	}
	fclose(gf1);
	FILE *gf2 = fopen("gaussian_9_9_9.txt", "r");
	n = R9*R9*R9;
	for (int i = 0; i < n; i++)
	{
		fscanf(gf2, "%g", &g9[i]);
	}
	fclose(gf2);
}

__global__ void d_compute_saliency()
{
	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;
	uint z = blockIdx.z*blockDim.z + threadIdx.z;
	printf("%d %d %d \n", x, y, z);
}

extern "C"
void gaussian(float *lch_volume, hipExtent volumeSize, float *out)
{
	auto len = volumeSize.width * volumeSize.height * volumeSize.depth;
	int w = volumeSize.width, h = volumeSize.height, d = volumeSize.depth;
	const int r1 = R5 / 2;
	const int r2 = R9 / 2;
	int w2 = w - r2;
	int h2 = h - r2;
	int d2 = d - r2;
	for (int z = r2; z < d2; z++)
	{
		for (int y = r2; y < h2; y++)
		{
			for (int x = r2; x < w2; x++)
			{
				int index = z*w*h + y*w + x;
				float sum9 = 0, sum5 = 0;
				for (int i = -r2; i <= r2; i++)
				{
					for (int j = -r2; j <= r2; j++)
					{
						for (int k = -r2; k <= r2; k++)
						{
							int idx = (z + i)*w*h + (y + j)*w + (x + k);
							sum9 += g9[(i + r2)*R9*R9 + (j + r2)*R9 + (k + r2)] * ((float *)lch_volume)[idx];
						}
					}
				}
				for (int i = -r1; i <= r1; i++)
				{
					for (int j = -r1; j <= r1; j++)
					{
						for (int k = -r1; k <= r1; k++)
						{
							int idx = (z + i)*w*h + (y + j)*w + (x + k);
							sum5 += g5[(i + r1)*R5*R5 + (j + r1)*R5 + (k + r1)] * ((float*)lch_volume)[idx];
						}
					}
				}
				out[index] = abs(sum5 - sum9);
			}
		}
	}
}

extern "C"
void compute_saliency()
{
	//std::cout << "compute_saliency()" << std::endl;
	dim3 blockSize3(16, 16, 16);
	dim3 gridSize3 = dim3(iDivUp(sizeOfVolume.width, blockSize3.x), iDivUp(sizeOfVolume.height, blockSize3.y), iDivUp(sizeOfVolume.depth, blockSize3.z));

	auto len = sizeOfVolume.width * sizeOfVolume.height * sizeOfVolume.depth;
	int w = sizeOfVolume.width, h = sizeOfVolume.height, d = sizeOfVolume.depth;
	memset(saliencyVolume, 0, sizeof(float) * len);

	float4 lch_array[BIN_COUNT] = { 0 };
	for (int i = 0; i < BIN_COUNT; i++)
	{
		float4 lch=rgb_to_lch(tf_array[i]);
		lch_array[i].x = lch.x;
		lch_array[i].y = lch.y;
		lch_array[i].z = lch.z;
		lch_array[i].w = lch.w;
	}

	float *lightness = (float *)malloc(len * sizeof(float));
	float *chroma = (float *)malloc(len * sizeof(float));
	float *g1 = (float *)malloc(len * sizeof(float));
	float *g2 = (float *)malloc(len * sizeof(float));
	memset(g1, 0, len * sizeof(float));
	memset(g2, 0, len * sizeof(float));

	//for (int z = 0; z < d; z++)
	//{
	//	for (int y = 0; y < h; y++)
	//	{
	//		for (int x = 0; x < w; x++)
	//		{
	//			int index = z*w*h + y*w + x;
	//			int intensity = (int)raw_volume[index];
	//			lightness[index] = lch_array[intensity].x;
	//			chroma[index] = lch_array[intensity].y;
	//		}
	//	}
	//}
	for (int i = 0; i < len; i++)
	{
		int intensity = (int)raw_volume[i];
		lightness[i] = lch_array[intensity].x;
		chroma[i] = lch_array[intensity].y;
	}

	gaussian(lightness, sizeOfVolume, g1);
	gaussian(chroma, sizeOfVolume, g2);

	//for (int z = 0; z < d; z++)
	//{
	//	for (int y = 0; y < h; y++)
	//	{
	//		for (int x = 0; x < w; x++)
	//		{
	//			int index = z*w*h + y*w + x;
	//			saliencyVolume[index] = (g1[index] + g2[index]) / 2;
	//			vwsVolume[index] = saliencyVolume[index] * visVolume[index];
	//		}
	//	}
	//}
	for (int i = 0; i < len; i++)
	{
		saliencyVolume[i] = (g1[i] + g2[i]) / 2;
		//vwsVolume[i] = saliencyVolume[i] * visVolume[i];
	}

	free(g1);
	free(g2);
	free(lightness);
	free(chroma);
}

extern "C"
void compute_vws()
{
	auto len = sizeOfVolume.width * sizeOfVolume.height * sizeOfVolume.depth;
	int w = sizeOfVolume.width, h = sizeOfVolume.height, d = sizeOfVolume.depth;
	memset(vwsVolume, 0, sizeof(float) * len);
	for (int i = 0; i < len; i++)
	{
		vwsVolume[i] = saliencyVolume[i] * visVolume[i];
	}
}

extern "C"
void compute_saliency_once()
{
	if (!saliency_once)
	{
		saliency_once = true;
		compute_saliency();
	}
}

extern "C"
void initCuda(void *h_volume, hipExtent volumeSize)
{
	auto len = volumeSize.width * volumeSize.height * volumeSize.depth;
	checkCudaErrors(hipMallocManaged(&raw_volume, sizeof(VolumeType) * len));
	memcpy(raw_volume, h_volume, sizeof(VolumeType) * len);
	//auto cube = malloc(sizeof(float) * len);
	//memset(cube, 0, sizeof(float) * len);
	//printf("%g\n", *((float*)cube+len-1));

	load_gaussians();
	checkCudaErrors(hipMallocManaged(&saliencyVolume, sizeof(float) * len));
	checkCudaErrors(hipMallocManaged(&vwsVolume, sizeof(float) * len));
	checkCudaErrors(hipMallocManaged(&featureVolume, sizeof(unsigned char) * len));

	compute_saliency();
	compute_vws();

	sizeOfVolume = volumeSize;
	printf("volumeSize \t %d %d %d\n", sizeOfVolume.width, sizeOfVolume.height, sizeOfVolume.depth);
	//printf("volumeSize \t %d %d %d\n", volumeSize.width, volumeSize.height, volumeSize.depth);

	//auto len = volumeSize.width * volumeSize.height * volumeSize.depth;
	checkCudaErrors(hipMallocManaged(&depthVolume, sizeof(float) * len));
	checkCudaErrors(hipMallocManaged(&visVolume, sizeof(float) * len));
	checkCudaErrors(hipMallocManaged(&countVolume, sizeof(int) * len));
	//printf("%g\n", *(visVolume + 1));
	//printf("%d\n", *(countVolume + 1));

	//auto tf2 = tf_array;
	//printf("sizeof \t histogram %d \t tf_array %d \t tf2 %d %d \n", sizeof(histogram) / sizeof(float), sizeof(tf_array) / sizeof(float4), sizeof(tf2), sizeof(float4));

	hipChannelFormatDesc channelDesc0 = hipCreateChannelDesc<VisibilityType>();
	//checkCudaErrors(hipMalloc3DArray(&d_visibilityArray, &channelDesc0, sizeOfVolume, hipArraySurfaceLoadStore));
	checkCudaErrors(hipMalloc3DArray(&d_visibilityArray, &channelDesc0, sizeOfVolume));
	//std::cout << "channelDesc0 \t" << channelDesc0.x << "\t" << channelDesc0.y << "\t" << channelDesc0.z << "\t" << channelDesc0.w << "\t" << channelDesc0.f << std::endl;

    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

	// set texture parameters
	volTex.normalized = true;                      // access with normalized texture coordinates
	//volTex.filterMode = hipFilterModeLinear;      // linear interpolation
	volTex.filterMode = hipFilterModePoint;      // nearest-neighbor interpolation
	volTex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	volTex.addressMode[1] = hipAddressModeClamp;

	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(volTex, d_volumeArray, channelDesc));

    // create transfer function texture
    float4 transferFunc[] =
    {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    //checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    //checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));
	checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(tf_array) / sizeof(float4), 1));
	checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, tf_array, sizeof(tf_array), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
	checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}

extern "C"
void freeCudaBuffers()
{
    checkCudaErrors(hipFreeArray(d_volumeArray));
	checkCudaErrors(hipFreeArray(d_transferFuncArray));
	checkCudaErrors(hipFreeArray(d_visibilityArray));
	checkCudaErrors(hipFree(depthVolume));
	checkCudaErrors(hipFree(visVolume));
	checkCudaErrors(hipFree(countVolume));
}

extern "C"
void render_visibility_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
	float density, float brightness, float transferOffset, float transferScale)
{
	auto len = sizeOfVolume.width * sizeOfVolume.height * sizeOfVolume.depth;
	//auto cube = malloc(sizeof(float) * len);
	//memset(visVolume, 0, sizeof(VisibilityType) * len);
	hipMemset(countVolume, 0, sizeof(int) * len);
	hipMemset(depthVolume, 0, sizeof(float) * len);
	hipMemset(visVolume, 0, sizeof(VisibilityType) * len);
	hipMemset(histogram, 0, sizeof(float)*BIN_COUNT);
	hipMemset(histogram2, 0, sizeof(float)*BIN_COUNT);

	//d_visibility << <gridSize, blockSize >> >(d_output, imageW, imageH, density, brightness, transferOffset, transferScale);
	//hipDeviceSynchronize();

	d_visibility<< <gridSize, blockSize >> >(d_output, imageW, imageH, density, brightness, transferOffset, transferScale);
	hipDeviceSynchronize();
}

extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale, int2 loc)
{
    //d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density, brightness, transferOffset, transferScale);

	auto len = sizeOfVolume.width * sizeOfVolume.height * sizeOfVolume.depth;
	//auto cube = malloc(sizeof(float) * len);
	//memset(visVolume, 0, sizeof(VisibilityType) * len);
	hipMemset(countVolume, 0, sizeof(int) * len);
	hipMemset(depthVolume, 0, sizeof(float) * len);
	hipMemset(visVolume, 0, sizeof(VisibilityType) * len);
	hipMemset(histogram, 0, sizeof(float)*BIN_COUNT);
	hipMemset(histogram2, 0, sizeof(float)*BIN_COUNT);

	//d_visibility << <gridSize, blockSize >> >(d_output, imageW, imageH, density, brightness, transferOffset, transferScale);
	//hipDeviceSynchronize();

	d_visibilityLocal << <gridSize, blockSize >> >(d_output, imageW, imageH, density, brightness, transferOffset, transferScale, loc);
	hipDeviceSynchronize();

	// copy data to 3D array
	hipMemcpy3DParms copyParams2 = { 0 };
	copyParams2.srcPtr = make_hipPitchedPtr(visVolume, sizeOfVolume.width * sizeof(VisibilityType), sizeOfVolume.width, sizeOfVolume.height);
	copyParams2.dstArray = d_visibilityArray;
	copyParams2.extent = sizeOfVolume;
	copyParams2.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams2));

	//checkCudaErrors(cudaMemcpyToArrayAsync(d_visibilityArray, 0, 0, countVolume, volumeSize.width*volumeSize.height*sizeof(int), hipMemcpyHostToDevice));

	// set texture parameters
	visTex.normalized = true;                      // access with normalized texture coordinates
	//visTex.filterMode = hipFilterModeLinear;      // linear interpolation
	visTex.filterMode = hipFilterModePoint;      // linear interpolation
	visTex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	visTex.addressMode[1] = hipAddressModeClamp;

	hipChannelFormatDesc channelDesc0 = hipCreateChannelDesc<VisibilityType>();
	checkCudaErrors(hipBindTextureToArray(visTex, d_visibilityArray, channelDesc0));

	d_renderVisibility << <gridSize, blockSize >> >(d_output, imageW, imageH, density, brightness, transferOffset, transferScale, loc);

	hipDeviceSynchronize();

	if (get_apply())
	{
		set_apply(false);
		printf("loc %d %d\n", loc.x, loc.y);
		blend_tf_rgba(make_float3(g_SelectedColor[0], g_SelectedColor[1], g_SelectedColor[2]));
	}

	if (get_gaussian())
	{
		set_gaussian(false);
		printf("loc %d %d\n", loc.x, loc.y);
		gaussian_tf(make_float3(g_SelectedColor[0], g_SelectedColor[1], g_SelectedColor[2]));
	}

	if (get_save())
	{
		set_save(false);

		char buffer[_MAX_PATH];
		sprintf(buffer, "~%s", volume_file);
		printf("save a visibility field and histograms to %s.\n", buffer);

		auto fp = fopen(buffer, "wb");
		fwrite(visVolume, sizeof(VisibilityType), len, fp);
		fclose(fp);

		{
			sprintf(buffer, "~%s.saliency.raw", volume_file);
			auto fp = fopen(buffer, "wb");
			fwrite(saliencyVolume, sizeof(float), len, fp);
			fclose(fp);
		}

		{
			sprintf(buffer, "~%s.vws.raw", volume_file);
			auto fp = fopen(buffer, "wb");
			fwrite(vwsVolume, sizeof(float), len, fp);
			fclose(fp);
		}

		{
			sprintf(buffer, "~%s.feature.raw", volume_file);
			auto fp = fopen(buffer, "wb");
			fwrite(featureVolume, sizeof(unsigned char), len, fp);
			fclose(fp);
		}

		sprintf(buffer, "~%s.depth.raw", volume_file);
		auto fp0 = fopen(buffer, "wb");
		fwrite(depthVolume, sizeof(float), len, fp0);
		fclose(fp0);

		sprintf(buffer, "~%s.data.txt", volume_file);
		auto fp6 = fopen(buffer, "w");
		for (int z = 0;z < sizeOfVolume.depth;z+=2)
		{
			for (int y = 0;y < sizeOfVolume.height;y+=2)
			{
				for (int x = 0;x < sizeOfVolume.width;x+=2)
				{
					int i = z*sizeOfVolume.width * sizeOfVolume.height + y*sizeOfVolume.width + x;
					fprintf(fp6, "{%f,%f,%f}\n", raw_volume[i] / 255.f, depthVolume[i], visVolume[i]);
				}
			}
		}
		fclose(fp6);

		sprintf(buffer, "~%s.txt", volume_file);
		auto fp1 = fopen(buffer, "w");
		for (int i = 0; i < BIN_COUNT; i++)
		{
			fprintf(fp1, "%f\n", histogram[i]);
		}
		fclose(fp1);

		printf("loc %d %d\n", loc.x, loc.y);
		sprintf(buffer, "~%s.2.txt", volume_file);
		auto fp2 = fopen(buffer, "w");
		for (int i = 0; i < BIN_COUNT; i++)
		{
			fprintf(fp2, "%f\n", histogram2[i]);
		}
		fclose(fp2);

		sprintf(buffer, "~%s.3.txt", volume_file);
		auto fp3 = fopen(buffer, "w");
		for (int i = 0; i < BIN_COUNT; i++)
		{
			fprintf(fp3, "%f\n", histogram3[i]);
		}
		fclose(fp3);

		sprintf(buffer, "~%s.4.txt", volume_file);
		auto fp4 = fopen(buffer, "w");
		for (int i = 0; i < BIN_COUNT; i++)
		{
			fprintf(fp4, "%f\n", histogram4[i]);
		}
		fclose(fp4);

		sprintf(buffer, "~%s.tf.txt", volume_file);
		auto fp5 = fopen(buffer, "w");
		for (int i = 0; i < BIN_COUNT; i++)
		{
			fprintf(fp5, "{%f,%f,%f,%f}\n", tf_array[i].x, tf_array[i].y, tf_array[i].z, tf_array[i].w);
		}
		fclose(fp5);
	}

	if (get_discard())
	{
		set_discard(false);
		restore_tf();
	}

	if (get_backup())
	{
		set_backup(false);
		backup_tf();
	}
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
